#include<iostream>
#include<hip/hip_runtime.h>
using namespace std;
#define N 10
__global__ void add(int *a,int *b,int *c){
    int tid=threadIdx.x;
    if(tid<N)
        c[tid]=a[tid]+b[tid];
}
int main(){
    int a[N],b[N],c[N];
    int *dev_a,*dev_b,*dev_c;
    hipMalloc(&dev_a,N*sizeof(int));
    hipMalloc(&dev_b,N*sizeof(int));
    hipMalloc(&dev_c,N*sizeof(int));
    
    for(int i=0;i<N;i++){
        a[i]=i;
        b[i]=i*i;
    }
    hipMemcpy(dev_a,a,N*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(dev_b,b,N*sizeof(int),hipMemcpyHostToDevice);
    add<<<1,N>>>(dev_a,dev_b,dev_c);//一个计算单元是一个kernel，即一个grid，我觉得第一个数是block数量，第二个是每一个blokc含有的thread数量
    hipMemcpy(c,dev_c,N*sizeof(int),hipMemcpyDeviceToHost);
    for(int i=0;i<N;i++){
        cout<<c[i]<<endl;

    }
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);
    return 0;
}