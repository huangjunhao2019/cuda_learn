#include<iostream>
#include<hip/hip_runtime.h>
using namespace std;
__global__ void multiply(int *a,int *b,int *devc){
    int i=blockIdx.x;
    devc[i]=a[i]*b[i];
//    return devc;
}
int main(){
    const int N=10;
    int *a,*b,*devc,*temp;
    temp=new int[N];
    hipMalloc(&a,N*sizeof(int));
    hipMalloc(&b,N*sizeof(int));
    hipMalloc(&devc,N*sizeof(int));

    for(int i=0;i<N;i++)
        temp[i]=i;
    hipMemcpy(a,temp,N*sizeof(int),hipMemcpyHostToDevice);

    for(int i=0;i<N;i++)
        temp[i]=2*i;
    hipMemcpy(b,temp,N*sizeof(int),hipMemcpyHostToDevice);

    multiply<<<N,1>>>(a,b,devc);
    int *result;
    result=new int[N];
    hipMemcpy(result,devc,sizeof(int)*N,hipMemcpyDeviceToHost);
    for(int i=0;i<N;i++)
        cout<<result[i]<<endl;
    hipFree(a);
    hipFree(b);
    hipFree(devc);
    return 0;
}