//本质上来说，几维的数组其实都是一维数组，不过是变变表现形式而已，二维数组加法没什么意思，
//就是一维数组加法，还是二维数组乘法有点意思
//这个方法，还不是高并发，高并发，应该是把求和那一块for也并发了。估计要用device，现在的并发度是4
#include<iostream>
#include<hip/hip_runtime.h>
using namespace std;
__global__ void mul(int *a,int *b,int *c){
    int row=blockIdx.x;
    int col=threadIdx.x;
    int temp_sum=0;
   for(int i=0;i<blockDim.x;i++){
        //temp_sum+=a[row][i]*b[i][row];
      temp_sum+=a[row*blockDim.x+i]*b[i*blockDim.x+col];
       // temp_sum+=row+col;
//       temp_sum=a[row*blockDim.x+col];
  }
   // c[row][col]=temp_sum;
   c[row*blockDim.x+col]=temp_sum;

}
int main(){
    int *a,*b,*dev_a,*dev_b,*dev_c,*c;
    const int N=2;
    a=new int[N*N];
    b=new int [N*N];
    c=new int[N*N];
    for(int i=0;i<N;i++){
        for(int j=0;j<N;j++){
            a[i*N+j]=i*N+j;
            b[i*N+j]=i*N+j;
            cout<<a[i*N+j]<<endl; 
        }
    }
    hipMalloc(&dev_a,N*N*sizeof(int));
    hipMalloc(&dev_b,N*N*sizeof(int));
    hipMalloc(&dev_c,N*N*sizeof(int));
    hipMemcpy(dev_a,a,N*N*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(dev_b,b,N*N*sizeof(int),hipMemcpyHostToDevice);

    mul<<<N,N>>>(dev_a,dev_b,dev_c);
    hipMemcpy(c,dev_c,N*N*sizeof(int),hipMemcpyDeviceToHost);
    for(int i=0;i<N;i++){
        for(int j=0;j<N;j++){
            cout<<c[i*N+j]<<" ";
        }
        cout<<endl;
    }
    return 0;

}