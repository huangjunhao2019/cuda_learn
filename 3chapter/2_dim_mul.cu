//本质上来说，几维的数组其实都是一维数组，不过是变变表现形式而已，二维数组加法没什么意思，
//就是一维数组加法，还是二维数组乘法有点意思
//这个方法，还不是高并发，高并发，应该是把求和那一块for也并发了。估计要用device，现在的并发度是4
#include<iostream>
#include<hip/hip_runtime.h>
using namespace std;
const int N=2;
__global__ void mul(int *a,int *b,int *c){//并发度为4的矩阵乘法
    int row=blockIdx.x;
    int col=threadIdx.x;
    int temp_sum=0;
   for(int i=0;i<blockDim.x;i++){
        //temp_sum+=a[row][i]*b[i][row];
      temp_sum+=a[row*blockDim.x+i]*b[i*blockDim.x+col];
       // temp_sum+=row+col;
//       temp_sum=a[row*blockDim.x+col];
  }
   // c[row][col]=temp_sum;
   c[row*blockDim.x+col]=temp_sum;

}
__global__ void mul_8(int *a,int *b,int *c){//实现了2*2*2个线程，每个都进行计算，并行度为8
    int row=blockIdx.x/N;
    int col=blockIdx.x%N;
    __shared__ int mul[N];
    //mul[0]=a[row][0]*b[0][col]
    mul[threadIdx.x]=a[row*N+threadIdx.x]*b[threadIdx.x*N+col];
    //mul[1]=a[row][1]*b[1][col]
    __syncthreads();
    int sum=0;
    for(int i=0;i<blockDim.x;i++)
        sum+=mul[i];
    c[blockIdx.x]=sum;
}
int main(){
    int *a,*b,*dev_a,*dev_b,*dev_c,*c;
    a=new int[N*N];
    b=new int [N*N];
    c=new int[N*N];
    for(int i=0;i<N;i++){
        for(int j=0;j<N;j++){
            a[i*N+j]=i*N+j;
            b[i*N+j]=i*N+j;
            cout<<a[i*N+j]<<endl; 
        }
    }
    hipMalloc(&dev_a,N*N*sizeof(int));
    hipMalloc(&dev_b,N*N*sizeof(int));
    hipMalloc(&dev_c,N*N*sizeof(int));
    hipMemcpy(dev_a,a,N*N*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(dev_b,b,N*N*sizeof(int),hipMemcpyHostToDevice);

   // mul<<<N,N>>>(dev_a,dev_b,dev_c);
   hipEvent_t start,stop;
   hipEventCreate(&start);
   hipEventCreate(&stop);
   hipEventRecord(start,0);

   mul_8<<<N*N,N>>>(dev_a,dev_b,dev_c);
   hipEventRecord(stop,0);
   hipEventSynchronize(stop);
   float elapsedTime;
   hipEventElapsedTime(&elapsedTime,start,stop);
   cout<<"Time: "<<elapsedTime<<endl;
    hipMemcpy(c,dev_c,N*N*sizeof(int),hipMemcpyDeviceToHost);
    for(int i=0;i<N;i++){
        for(int j=0;j<N;j++){
            cout<<c[i*N+j]<<" ";
        }
        cout<<endl;
    }
    return 0;

}