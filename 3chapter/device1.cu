#include<iostream>
#include<hip/hip_runtime.h>
using namespace std;
int main(){
    hipDeviceProp_t prop;

    int count;
    hipGetDeviceCount(&count);
    for(int i=0;i<count;i++){
        hipGetDeviceProperties(&prop,i);
        cout<<"--- General Information for device "<<i<<endl;
        cout<<"Name: "<<prop.name<<endl;
        cout<<"Compute capability: "<<prop.major<<", "<<prop.minor<<endl;
        cout<<"Clock rate: "<<prop.clockRate<<endl;
        cout<<"Device copy overlap: "<<prop.deviceOverlap<<endl;
        cout<<"Total global memory: "<<prop.totalGlobalMem<<endl;
        cout<<"Total const memory: "<<prop.totalConstMem<<endl;
        return 0;
    }
}