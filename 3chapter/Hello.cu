
#include <hip/hip_runtime.h>
#include<iostream>
using namespace std;
__global__ void add(int a,int b,int *c){
    *c=a+b;
}
int main()
{
    int c;
    int *dev_c;
    hipMalloc(&dev_c,sizeof(int));
    add<<<1,1>>>(2,7,dev_c);
    hipMemcpy(&c,dev_c,sizeof(int),hipMemcpyDeviceToHost);
    cout<<"2+7="<<c<<endl;
    hipFree(dev_c);
    return 0;
}