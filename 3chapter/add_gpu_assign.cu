#include<iostream>
#include<hip/hip_runtime.h>
using namespace std;
__global__ void assign(int *a,int N){
 //using gpu to assign value for a  
    int i=blockIdx.x;
    if(i<N)
        a[i]=i*i;
}
int main(){
    const int N=10;
    int *a,*dev_a,*b,*dev_b;
    a=new int[N];
    b=new int[N];
    hipMalloc(&dev_a,N*sizeof(int));
    hipMalloc(&dev_b,N*sizeof(int));
    assign<<<N,1>>>(dev_a,N);
    assign<<<N,1>>>(dev_b,N);
    hipMemcpy(a,dev_a,N*sizeof(int),hipMemcpyDeviceToHost);
    hipMemcpy(b,dev_b,N*sizeof(int),hipMemcpyDeviceToHost);
    for(int i=0;i<N;i++)
        cout<<a[i]<<endl;
    for(int i=0;i<N;i++)
        cout<<b[i]<<endl;
    hipFree(dev_a);
    hipFree(dev_b);
    return 0;

}