#include<iostream>
#include<hip/hip_runtime.h>
using namespace std;
//这种__shared__其实没什么，就是一个block内的线程能够维护一个全局变量
#define imin(a,b) (a<b?a:b)
const int N=33*1024;
const int threadsPerBlock=256;
const int blocksPerGrid=imin(32,(N+threadsPerBlock-1)/threadsPerBlock);
__global__ void dot(float *a,float *b,float *c){
    __shared__ float cache[threadsPerBlock];
    int tid=threadIdx.x+blockIdx.x*blockDim.x;
    int cacheIndex=threadIdx.x;

    float temp=0;
    while(tid<N){
        temp+=a[tid]*b[tid];
        tid+=blockDim.x*gridDim.x;
    }
    cache[cacheIndex]=temp;
    __syncthreads();

    float sum=0;
    for(int i=0;i<threadsPerBlock;i++)
        sum+=cache[i];
    c[blockIdx.x]=sum;
}
int main(){
    float *a,*b,*c;
    float *dev_a,*dev_b,*dev_c;
    a=new float[N*sizeof(float)];
    b=new float[N*sizeof(float)];
    c=new float[N*sizeof(float)];

    hipMalloc(&dev_a,N*sizeof(float));
    hipMalloc(&dev_b,N*sizeof(float));
    hipMalloc(&dev_c,N*sizeof(float));

    for(int i=0;i<N;i++){
        a[i]=i;
        b[i]=i*2;
    }
    
    hipMemcpy(dev_a,a,N*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(dev_b,b,N*sizeof(float),hipMemcpyHostToDevice);
    dot<<<blocksPerGrid,threadsPerBlock>>>(dev_a,dev_b,dev_c);
    hipMemcpy(c,dev_c,N*sizeof(float),hipMemcpyDeviceToHost);
    float sum=0;
    for(int i=0;i<blocksPerGrid;i++)
        sum+=c[i];
    cout<<"Summary="<<sum<<endl;
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);
    free(a);
    free(b);
    free(c);
    return 0;
}