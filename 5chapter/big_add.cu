#include<iostream>
#include<hip/hip_runtime.h>
using namespace std;
#define N 40*1024
__global__ void add(int *a,int *b,int *c){
    int tid=threadIdx.x+blockIdx.x*blockDim.x;
    while(tid<N){
        c[tid]=a[tid]+b[tid];
        tid+=blockDim.x*gridDim.x;
    }
}
int main(){
    int a[N],b[N],c[N];
    int *dev_a,*dev_b,*dev_c;
    hipMalloc(&dev_a,N*sizeof(int));
    hipMalloc(&dev_b,N*sizeof(int));
    hipMalloc(&dev_c,N*sizeof(int));
    for(int i=0;i<N;i++){
        a[i]=i;
        b[i]=i*i;
    }
    hipMemcpy(dev_a,a,N*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(dev_b,b,N*sizeof(int),hipMemcpyHostToDevice);
    add<<<128,128>>>(dev_a,dev_b,dev_c);
    hipMemcpy(c,dev_c,N*sizeof(int),hipMemcpyDeviceToHost);
    for(int i=0;i<N;i++){
        if(i!=0&&i%20==0)
            cout<<c[i]<<endl;
        else
            cout<<c[i]<<" ";
    }
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);
    return 0;
}