#include<iostream>
#include<hip/hip_runtime.h>
using namespace std;
__global__ void get_block(int *c){
    c[0]=blockDim.x;
    c[1]=gridDim.x;
}
int main(){
    int c[2];
    int *dev_c;
    hipMalloc(&dev_c,2*sizeof(int));
    get_block<<<10,10>>>(dev_c);
    hipMemcpy(c,dev_c,2*sizeof(int),hipMemcpyDeviceToHost);
    for(int i=0;i<2;i++)
        cout<<c[i]<<endl;
  //  cout<<"threadPerBlock="<<threadsPerBlock<<endl;
  //  cout<<"blocksPerGrid="<<blocksPerGrid<<endl; 16，17两个cout均显示相关变量没有定义，这说明这两个变量就是一般变量，
  //需要先声明，在使用
    return 0;
}