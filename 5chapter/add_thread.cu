#include<iostream>
#include<hip/hip_runtime.h>
using namespace std;
#define N 10
__global__ void add(int *a,int *b,int *c){
    int tid=threadIdx.x;
    if(tid<N)
        c[tid]=a[tid]+b[tid];
}
int main(){
    int a[N],b[N],c[N];
    int *dev_a,*dev_b,*dev_c;
    hipMalloc(&dev_a,N*sizeof(int));
    hipMalloc(&dev_b,N*sizeof(int));
    hipMalloc(&dev_c,N*sizeof(int));

    for(int i=0;i<N;i++){
        a[i]=i;
        b[i]=i*i;
    }
    hipMemcpy(dev_a,a,N*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(dev_b,b,N*sizeof(int),hipMemcpyHostToDevice);
    add<<<1,N>>>(dev_a,dev_b,dev_c);
    hipMemcpy(c,dev_c,N*sizeof(int),hipMemcpyDeviceToHost);
    for(int i=0;i<N;i++)
        cout<<a[i]<<" "<<b[i]<<" "<<c[i]<<endl;
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    return 0;
}