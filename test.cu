#include<iostream>
#include<hip/hip_runtime.h>
using namespace std;
__global__ void add(int *a,const int *b){
    int i=blockIdx.x;
    a[i]+=b[i];
}
int main(){
    const int N=10;
    int *a,*b,*temp;

    temp=new int[N];
    
    hipMalloc(&a,N*sizeof(int));
    hipMalloc(&b,N*sizeof(int));

    for(int i=0;i<N;i++)
        temp[i]=i;
    
    hipMemcpy(a,temp,N*sizeof(int),hipMemcpyHostToDevice);
    
    for(int i=0;i<N;i++)
        temp[i]=2*i;
    hipMemcpy(b,temp,N*sizeof(int),hipMemcpyHostToDevice);
    add<<<N,1>>>(a,b);

    hipMemcpy(temp,a,N*sizeof(int),hipMemcpyDeviceToHost);
    for(int i=0;i<N;i++)
        cout<<temp[i]<<endl;
    delete[] temp;
    hipFree(a);
    hipFree(b);


    
}
